#include "hip/hip_runtime.h"
#define DEBUG(...) if(threadIdx.x == 0 && blockIdx.x == 0) printf(__VA_ARGS__)

struct bfloat4 {
	__hip_bfloat16 x, y, z, w;
};

__global__ void tonemap(const bfloat4* bins, uchar4* image, unsigned int dims_x, unsigned int dims_y, float gamma, float scale_constant, float brightness, float vibrancy) {

	//DEBUG("%dx%d (%f,%f,%f,%f)\n", dims_x, dims_y, gamma, scale_constant, brightness, vibrancy);

	const uchar4 background = { 0, 255, 0, 255 };
	uint2 pos = {
		threadIdx.x + blockIdx.x * blockDim.x,
		threadIdx.y + blockIdx.y * blockDim.y
	};

	if (pos.x >= dims_x || pos.y >= dims_y) return;

	unsigned int bin_idx = (pos.y) * dims_x + pos.x;
	bfloat4& col_ref = bins[bin_idx];
	float4 col = {col_ref.x, col_ref.y, col_ref.z, col_ref.w};

	DEBUG("%f %f %f %f\n", col.x, col.y, col.z, col.w);

	if(col.w == 0.0) {
		image[bin_idx] = background;
		return;
	}

	col.w += 1;
	float factor = (col.w == 0.0f)? 0.0f : 0.5f * brightness * logf(1.0f + col.w * scale_constant) * 0.434294481903251827651128918916f / (col.w);
	//col.x *= factor; col.y *= factor; col.z *= factor; col.w *= factor;

	float inv_gamma = 1.0f / gamma;
	float z = pow(col.w, inv_gamma);
	float gamma_factor = z / col.w;

	col.x *= gamma_factor; 
	col.y *= gamma_factor;
	col.z *= gamma_factor;
	//col.w *= gamma_factor;

	#define interp(left, right, mix) ((left) * (1.0f - (mix)) + (right) * (mix))

	image[bin_idx] = {
		(unsigned char) min(255.0f, col.x * 255.0f),
		(unsigned char) min(255.0f, col.y * 255.0f),
		(unsigned char) min(255.0f, col.z * 255.0f),
		255//(unsigned char) min(255.0f, col.w * 255.0f)
	};
	
	/*col.w = max(0.0, min(col.w, 1.0f));
	image[bin_idx] = {
		(unsigned char) interp(background.x, min(255.0f, col.x * 255.0f), col.w),
		(unsigned char) interp(background.y, min(255.0f, col.y * 255.0f), col.w),
		(unsigned char) interp(background.z, min(255.0f, col.z * 255.0f), col.w),
		(unsigned char) 255
	};*/
}